#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
namespace Thrust {

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata, float *time) {
    thrust::device_vector<int> ivec(idata, idata+n);
    thrust::device_vector<int> ovec(odata, odata+n);

        hipEvent_t begin, end;
        hipEventCreate(&begin);
        hipEventCreate(&end);
        hipEventRecord(begin, 0);

    thrust::exclusive_scan(ivec.begin(), ivec.end(), ovec.begin());

        hipEventRecord(end, 0);
        hipEventSynchronize(end);
        hipEventElapsedTime(time, begin, end);
        hipEventDestroy(begin);
        hipEventDestroy(end);

    thrust::copy(ovec.begin(), ovec.end(), odata);
}

}
}
