#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
namespace Efficient {

__global__ void kUpSweep(int d, int *data) {
    int k = threadIdx.x;
    int exp_d  = (int)exp2f(d);
    int exp_d1 = (int)exp2f(d+1);
    if (k % exp_d1 == 0) {
        data[k + exp_d1 - 1] += data[k + exp_d - 1];
    }
}

__global__ void kDownSweep(int d, int *data) {
    int k = threadIdx.x;
    if (k % (int)exp2f(d+1) == 0) {
        int left  = k + (int)exp2f(d) - 1;
        int right = k + (int)exp2f(d+1) - 1;
        int t = data[left];
        data[left] = data[right];
        data[right] += t;
    }
}

/*
 * In-place scan on `dev_idata`, which must be a device memory pointer.
 */
void dv_scan(int n, int *dev_idata) {
    for (int d = 0; d < ilog2ceil(n)-1; d++) {
        kUpSweep<<<1, n>>>(d, dev_idata);
    }

    int z = 0;
    hipMemcpy(&dev_idata[n-1], &z, sizeof(int), hipMemcpyHostToDevice);

    for (int d = ilog2ceil(n)-1; d >= 0; d--) {
        kDownSweep<<<1, n>>>(d, dev_idata);
    }
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int size, int *odata, const int *input) {
    int *idata;
    int n;

    if (size & (size-1) != 0) { // if size is not a power of 2
        n = (int)exp2f(ilog2ceil(size));
        idata = (int*)malloc(n * sizeof(int));
        memcpy(idata, input, n * sizeof(int));
        for (int j = size; j < n; j++) {
            idata[j] = 0;
        }
    } else {
        n = size;
        idata = (int*)malloc(n * sizeof(int));
        memcpy(idata, input, n * sizeof(int));
    }

    int *dv_idata;
    int array_size = n * sizeof(int);

    hipMalloc((void**) &dv_idata, array_size);
    hipMemcpy(dv_idata, idata, array_size, hipMemcpyHostToDevice);

    dv_scan(n, dv_idata);

    hipMemcpy(odata, dv_idata, array_size, hipMemcpyDeviceToHost);
    hipFree(dv_idata);
}

__global__ void kScatter(int n, int *odata, int *indices, int *idata) {
    int k = threadIdx.x;
    if (k >= n) { return; }
    if (k == n-1) {
        // always take the last element
        // `compact` will adjust size appropriately
        odata[indices[k]] = idata[k];
    } else if (indices[k] != indices[k+1]) {
        odata[indices[k]] = idata[k];
    }
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int size, int *odata, const int *input) {
    int *idata;
    int n;

    if (size & (size-1) != 0) { // if size is not a power of 2
        n = (int)exp2f(ilog2ceil(size));
        idata = (int*)malloc(n * sizeof(int));
        memcpy(idata, input, n * sizeof(int));
        for (int j = size; j < n; j++) {
            idata[j] = 0;
        }
    } else {
        n = size;
        idata = (int*)malloc(n * sizeof(int));
        memcpy(idata, input, n * sizeof(int));
    }

    int *dev_indices;
    int *dev_odata;
    int *dev_idata;
    int array_size = n * sizeof(int);

    hipMalloc((void**) &dev_indices, array_size);
    hipMalloc((void**) &dev_odata, array_size);

    hipMalloc((void**) &dev_idata, array_size);
    hipMemcpy(dev_idata, idata, array_size, hipMemcpyHostToDevice);

    StreamCompaction::Common::kernMapToBoolean<<<1, n>>>(n, dev_indices, dev_idata);

    int last;
    hipMemcpy(&last, dev_indices + n-1, sizeof(int), hipMemcpyDeviceToHost);

    dv_scan(n, dev_indices);
    int streamSize;
    hipMemcpy(&streamSize, dev_indices + n-1, sizeof(int), hipMemcpyDeviceToHost);

    kScatter<<<1, n>>>(n, dev_odata, dev_indices, dev_idata);
    hipMemcpy(odata, dev_odata, array_size, hipMemcpyDeviceToHost);

    // The kernel always copies the last elt.
    // Adjust the size to include it if desired.
    if (last == 1) {
        streamSize++;
    }

    return streamSize;
}

}
}
