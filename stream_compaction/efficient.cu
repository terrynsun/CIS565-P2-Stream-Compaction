#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
namespace Efficient {

int BLOCK_SIZE = (2 << 7);

__global__ void kUpSweep(int d, int *data) {
    int k = (blockDim.x * blockIdx.x) + threadIdx.x;
    int exp_d  = (int)exp2f(d);
    int exp_d1 = (int)exp2f(d+1);
    if (k % exp_d1 == 0) {
        data[k + exp_d1 - 1] += data[k + exp_d - 1];
    }
}

__global__ void kDownSweep(int d, int *data) {
    int k = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (k % (int)exp2f(d+1) == 0) {
        int left  = k + (int)exp2f(d) - 1;
        int right = k + (int)exp2f(d+1) - 1;
        int t = data[left];
        data[left] = data[right];
        data[right] += t;
    }
}

/*
 * In-place scan on `dev_idata`, which must be a device memory pointer.
 */
void dv_scan(int n, int *dev_idata) {
    int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

    for (int d = 0; d < ilog2ceil(n)-1; d++) {
        kUpSweep<<<numBlocks, BLOCK_SIZE>>>(d, dev_idata);
        checkCUDAError("scan");
    }

    int z = 0;
    hipMemcpy(&dev_idata[n-1], &z, sizeof(int), hipMemcpyHostToDevice);

    for (int d = ilog2ceil(n)-1; d >= 0; d--) {
        kDownSweep<<<numBlocks, BLOCK_SIZE>>>(d, dev_idata);
        checkCUDAError("scan");
    }
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int size, int *odata, const int *input, float *time, int blockSize) {
    BLOCK_SIZE = blockSize;
    int *idata;
    int n;

    if (size & (size-1) != 0) { // if size is not a power of 2
        n = (int)exp2f(ilog2ceil(size));
        idata = (int*)malloc(n * sizeof(int));
        memcpy(idata, input, n * sizeof(int));
        for (int j = size; j < n; j++) {
            idata[j] = 0;
        }
    } else {
        n = size;
        idata = (int*)malloc(n * sizeof(int));
        memcpy(idata, input, n * sizeof(int));
    }


    int array_size = n * sizeof(int);
    int *dv_idata;

    hipMalloc((void**) &dv_idata, array_size);
    hipMemcpy(dv_idata, idata, array_size, hipMemcpyHostToDevice);

        hipEvent_t begin, end;
        hipEventCreate(&begin);
        hipEventCreate(&end);
        hipEventRecord(begin, 0);

    dv_scan(n, dv_idata);

        hipEventRecord(end, 0);
        hipEventSynchronize(end);
        hipEventElapsedTime(time, begin, end);
        hipEventDestroy(begin);
        hipEventDestroy(end);

    hipMemcpy(odata, dv_idata, array_size, hipMemcpyDeviceToHost);
    hipFree(dv_idata);
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int size, int *odata, const int *input, float *time, int blockSize) {
    BLOCK_SIZE = blockSize;

    int *idata;
    int n;

    if (size & (size-1) != 0) { // if size is not a power of 2
        n = (int)exp2f(ilog2ceil(size));
        idata = (int*)malloc(n * sizeof(int));
        memcpy(idata, input, n * sizeof(int));
        for (int j = size; j < n; j++) {
            idata[j] = 0;
        }
    } else {
        n = size;
        idata = (int*)malloc(n * sizeof(int));
        memcpy(idata, input, n * sizeof(int));
    }

    int *dev_indices;
    int *dev_odata;
    int *dev_idata;
    int array_size = n * sizeof(int);
    int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hipMalloc((void**) &dev_indices, array_size);
    hipMalloc((void**) &dev_odata, array_size);

    hipMalloc((void**) &dev_idata, array_size);
    hipMemcpy(dev_idata, idata, array_size, hipMemcpyHostToDevice);

        hipEvent_t begin, end;
        hipEventCreate(&begin);
        hipEventCreate(&end);
        hipEventRecord(begin, 0);

    StreamCompaction::Common::kernMapToBoolean<<<numBlocks, blockSize>>>(n, dev_indices, dev_idata);

    int last;
    hipMemcpy(&last, dev_indices + n-1, sizeof(int), hipMemcpyDeviceToHost);

    dv_scan(n, dev_indices);
    int streamSize;
    hipMemcpy(&streamSize, dev_indices + n-1, sizeof(int), hipMemcpyDeviceToHost);

    StreamCompaction::Common::kernScatter<<<numBlocks, blockSize>>>(n, dev_odata, dev_indices, dev_idata);

        hipEventRecord(end, 0);
        hipEventSynchronize(end);
        hipEventElapsedTime(time, begin, end);
        hipEventDestroy(begin);
        hipEventDestroy(end);

    hipMemcpy(odata, dev_odata, array_size, hipMemcpyDeviceToHost);

    // The kernel always copies the last elt.
    // Adjust the size to include it if desired.
    if (last == 1) {
        streamSize++;
    }

    return streamSize;
}

}
}
