#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
namespace Naive {

int BLOCK_SIZE = 128;

__global__ void kScan(int d, int *odata, const int *idata) {
    int k = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (k >= (int)exp2f(d-1)) {
        odata[k] = idata[k - (int)exp2f(d-1)] + idata[k];
    } else {
        odata[k] = idata[k];
    }
}

__global__ void kShift(int n, int *odata, int *idata) {
    int k = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (k >= n) { return; }
    if (k == 0) {
        odata[0] = 0;
    } else {
        odata[k] = idata[k-1];
    }
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
__host__ void scan(int n, int *odata, const int *idata) {
    int array_size = n * sizeof(int);
    int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

    int *A;
    int *B;

    hipMalloc((void**) &A, array_size);
    hipMalloc((void**) &B, array_size);
    hipMemcpy(A, idata, array_size, hipMemcpyHostToDevice);

    int *in;
    int *out;

    for (int d = 1; d < ilog2ceil(n)+1; d++) {
        in  = (d % 2 == 1) ? A : B;
        out = (d % 2 == 1) ? B : A;
        kScan<<<numBlocks, BLOCK_SIZE>>>(d, out, in);
        checkCUDAError("scan");
    }

    // shift odata to the right for exclusive scan
    kShift<<<numBlocks, BLOCK_SIZE>>>(n, in, out);

    hipMemcpy(odata, in, array_size, hipMemcpyDeviceToHost);

    hipFree(A);
    hipFree(B);
}

}
}
