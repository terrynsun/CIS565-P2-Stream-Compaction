#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
namespace Shared {

int BLOCK_SIZE = 4;

__global__ void kUpSweep(int *data, int n) {
    int t = threadIdx.x;
    int start = blockDim.x * blockIdx.x;

    // Load into shared memory
    extern __shared__ int shared[];
    shared[t] = data[start+t];
    __syncthreads();

    n = blockDim.x;
    int iters = ilog2ceil(n);

    for (int d = 0; d < iters; d++) {
        int exp_d1 = (int)exp2f(d+1);
        int k = t * exp_d1;

        if (k < n && k + exp_d1 - 1 < n) {
            int exp_d  = (int)exp2f(d);
            shared[k + exp_d1 - 1] += shared[k + exp_d - 1];
        }
        __syncthreads();
    }

    // Load back into global memory
    data[start+t] = shared[t];
    __syncthreads();
}

__global__ void kStoreZero(int *data, int count, int blockSize) {
    int t = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (t < count) {
        int index = (t+1)*blockSize - 1;
        data[index] = 0;
    }
}

__global__ void kDownSweep(int *data, int n) {
    int t = threadIdx.x;
    int start = blockDim.x * blockIdx.x;

    // Load into shared memory
    extern __shared__ int shared[];
    shared[t] = data[start+t];
    __syncthreads();

    n = blockDim.x;
    int iters = ilog2ceil(n)-1;
    for (int d = iters; d >= 0; d--) {

        int k = t * (int)exp2f(d+1);
        int left  = k + (int)exp2f(d)   - 1;
        int right = k + (int)exp2f(d+1) - 1;

        if (k < n && right < n) {
            int left_data  = shared[left];

            shared[left]   = shared[right];
            shared[right] += left_data;
        }
        __syncthreads();
    }

    // Load back into global memory
    if (t < n) {
        data[start+t] = shared[t];
    }
    __syncthreads();
}

__global__ void kCollectBlockSums(int *odata, int *idata,
        int numBlocks, int blockSize) {
    int t = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (t < numBlocks) {
        int index = (t+1)*blockSize - 1;
        odata[t] = idata[index];
    }
}

__global__ void kAddRunningBlockTotal(int *data, int *totals) {
    int idx = (blockDim.x * blockIdx.x) + threadIdx.x;
    data[idx] += totals[blockIdx.x];
}

// Shift left
// data[t-1] = data[t]; OR data[t] = data[t+1];
// data[n-1] = data[n-1] + last[blockDim.x];
__global__ void kConvertExToInclusive(int *data, int *lastData, int n) {
    int idx = (blockDim.x * blockIdx.x) + threadIdx.x;
    int t = threadIdx.x;
    if (t < blockDim.x - 1) {
        int val = data[idx+1];
        __syncthreads();
        data[idx] = val;
    } else if (t == blockDim.x - 1) {
        int val = data[idx];
        __syncthreads();
        int last = lastData[blockIdx.x];
        data[idx] = val + last;
    } else {
        __syncthreads();
    }
}

__global__ void kConvertInToExclusive(int *data, int n) {
    int t = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (t < n) {
        int val = data[t];
        __syncthreads();
        data[t+1] = val;
    } else {
        __syncthreads();
    }
    data[0] = 0;
}

/*
 * In-place scan on `dev_data`, which must be a device memory pointer.
 */
void dv_scan(int n, int *dev_data) {
    // Number of blocks of data
    int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    // Number of blocks, when operating on the set of blocks
    int numBlocksForBlocks = (n + numBlocks - 1) / numBlocks;

    int *blockTotals;
    hipMalloc((void**) &blockTotals, n*sizeof(int));

    // Store last value, to convert into inclusive scan.
    kCollectBlockSums<<<numBlocksForBlocks, numBlocks>>>(
            blockTotals, dev_data, numBlocks, BLOCK_SIZE);

    // Run INCLUSIVE scan on each block
    kUpSweep<<<numBlocks, BLOCK_SIZE, BLOCK_SIZE*sizeof(int)>>>(dev_data, n);
    kStoreZero<<<numBlocksForBlocks, numBlocks>>>(dev_data, numBlocks, BLOCK_SIZE);
    kDownSweep<<<numBlocks, BLOCK_SIZE, BLOCK_SIZE*sizeof(int)>>>(dev_data, n);
    kConvertExToInclusive<<<numBlocks, BLOCK_SIZE, BLOCK_SIZE*sizeof(int)>>>(
            dev_data, blockTotals, n);

    // Build array of sums from each block
    kCollectBlockSums<<<numBlocksForBlocks, numBlocks>>>(
            blockTotals, dev_data, numBlocks, BLOCK_SIZE);

    // Find block increments (EXclusive scan)
    kUpSweep<<<1, numBlocks, numBlocks*sizeof(int)>>>(blockTotals, numBlocks);
    kStoreZero<<<1, 1>>>(blockTotals, 1, numBlocks);
    kDownSweep<<<1, numBlocks, numBlocks*sizeof(int)>>>(blockTotals, numBlocks);

    // Add block increments back into each blocks
    kAddRunningBlockTotal<<<numBlocks, BLOCK_SIZE>>>(dev_data, blockTotals);

    kConvertInToExclusive<<<numBlocks, BLOCK_SIZE, BLOCK_SIZE*sizeof(int)>>>(
            dev_data, n);

    checkCUDAError("scan");
    hipFree(blockTotals);
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int size, int *odata, const int *input) {
    int *idata;
    int n;

    if (size & (size-1) != 0) { // if size is not a power of 2
        n = (int)exp2f(ilog2ceil(size));
        idata = (int*)malloc(n * sizeof(int));
        memcpy(idata, input, n * sizeof(int));
        for (int j = size; j < n; j++) {
            idata[j] = 0;
        }
    } else {
        n = size;
        idata = (int*)malloc(n * sizeof(int));
        memcpy(idata, input, n * sizeof(int));
    }

    int array_size = n * sizeof(int);
    int *dv_idata;

    hipMalloc((void**) &dv_idata, array_size);
    hipMemcpy(dv_idata, idata, array_size, hipMemcpyHostToDevice);

    dv_scan(n, dv_idata);

    hipMemcpy(odata, dv_idata, array_size, hipMemcpyDeviceToHost);
    hipFree(dv_idata);
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int size, int *odata, const int *input) {
    int *idata;
    int n;

    if (size & (size-1) != 0) { // if size is not a power of 2
        n = (int)exp2f(ilog2ceil(size));
        idata = (int*)malloc(n * sizeof(int));
        memcpy(idata, input, n * sizeof(int));
        for (int j = size; j < n; j++) {
            idata[j] = 0;
        }
    } else {
        n = size;
        idata = (int*)malloc(n * sizeof(int));
        memcpy(idata, input, n * sizeof(int));
    }

    int *dev_indices;
    int *dev_odata;
    int *dev_idata;
    int array_size = n * sizeof(int);
    int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hipMalloc((void**) &dev_indices, array_size);
    hipMalloc((void**) &dev_odata, array_size);

    hipMalloc((void**) &dev_idata, array_size);
    hipMemcpy(dev_idata, idata, array_size, hipMemcpyHostToDevice);

    StreamCompaction::Common::kernMapToBoolean<<<numBlocks, BLOCK_SIZE>>>(n, dev_indices, dev_idata);

    int last;
    hipMemcpy(&last, dev_indices + n-1, sizeof(int), hipMemcpyDeviceToHost);

    dv_scan(n, dev_indices);
    int streamSize;
    hipMemcpy(&streamSize, dev_indices + n-1, sizeof(int), hipMemcpyDeviceToHost);

    StreamCompaction::Common::kernScatter<<<numBlocks, BLOCK_SIZE>>>(n, dev_odata, dev_indices, dev_idata);

    hipMemcpy(odata, dev_odata, array_size, hipMemcpyDeviceToHost);

    // The kernel always copies the last elt.
    // Adjust the size to include it if desired.
    if (last == 1) {
        streamSize++;
    }

    return streamSize;
}

}
}
