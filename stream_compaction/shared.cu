#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
namespace Shared {

int BLOCK_SIZE = 1<<8;

__global__ void kUpSweep(int *data, int n) {
    int t = (blockDim.x * blockIdx.x) + threadIdx.x;

    int iters = ilog2ceil(n)-1;

    // Load into shared memory
    extern __shared__ int shared[];
    shared[t] = data[t];
    __syncthreads();

    for (int d = 0; d < iters; d++) {
        int exp_d1 = (int)exp2f(d+1);
        int k = t * exp_d1;

        if (k < n && k + exp_d1 - 1 < n) {
            int exp_d  = (int)exp2f(d);
            shared[k + exp_d1 - 1] += shared[k + exp_d - 1];
        }
        __syncthreads();
    }

    // Load back into global memory
    data[t] = shared[t];
    __syncthreads();

}

__global__ void kDownSweep(int *data, int n) {
    int t = (blockDim.x * blockIdx.x) + threadIdx.x;
    int iters = ilog2ceil(n)-1;

    // Load into shared memory
    extern __shared__ int shared[];
    shared[t] = data[t];
    __syncthreads();

    for (int d = iters; d >= 0; d--) {
        int k = t * (int)exp2f(d+1);
        int left  = k + (int)exp2f(d) - 1;
        int right = k + (int)exp2f(d+1) - 1;
        if (k < n && right < n) {
            int left_data  = data[left];
            shared[left]   = shared[right];
            shared[right] += left_data;
        }
        __syncthreads();
    }

    // Load back into global memory
    if (t < n) {
        data[t] = shared[t];
    }
    __syncthreads();
}

/*
 * In-place scan on `dev_data`, which must be a device memory pointer.
 */
void dv_scan(int n, int *dev_data) {
    int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    kUpSweep<<<numBlocks, BLOCK_SIZE, n*sizeof(int)>>>(dev_data, n);
    checkCUDAError("upsweep");

    int z = 0;
    hipMemcpy(&dev_data[n-1], &z, sizeof(int), hipMemcpyHostToDevice);
    checkCUDAError("cudamemcpy");

    kDownSweep<<<numBlocks, BLOCK_SIZE, n*sizeof(int)>>>(dev_data, n);
    checkCUDAError("downsweep");
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int size, int *odata, const int *input) {
    int *idata;
    int n;

    if (size & (size-1) != 0) { // if size is not a power of 2
        n = (int)exp2f(ilog2ceil(size));
        idata = (int*)malloc(n * sizeof(int));
        memcpy(idata, input, n * sizeof(int));
        for (int j = size; j < n; j++) {
            idata[j] = 0;
        }
    } else {
        n = size;
        idata = (int*)malloc(n * sizeof(int));
        memcpy(idata, input, n * sizeof(int));
    }


    int array_size = n * sizeof(int);
    int *dv_idata;

    hipMalloc((void**) &dv_idata, array_size);
    hipMemcpy(dv_idata, idata, array_size, hipMemcpyHostToDevice);

    dv_scan(n, dv_idata);

    hipMemcpy(odata, dv_idata, array_size, hipMemcpyDeviceToHost);
    hipFree(dv_idata);
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int size, int *odata, const int *input) {
    int *idata;
    int n;

    if (size & (size-1) != 0) { // if size is not a power of 2
        n = (int)exp2f(ilog2ceil(size));
        idata = (int*)malloc(n * sizeof(int));
        memcpy(idata, input, n * sizeof(int));
        for (int j = size; j < n; j++) {
            idata[j] = 0;
        }
    } else {
        n = size;
        idata = (int*)malloc(n * sizeof(int));
        memcpy(idata, input, n * sizeof(int));
    }

    int *dev_indices;
    int *dev_odata;
    int *dev_idata;
    int array_size = n * sizeof(int);
    int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hipMalloc((void**) &dev_indices, array_size);
    hipMalloc((void**) &dev_odata, array_size);

    hipMalloc((void**) &dev_idata, array_size);
    hipMemcpy(dev_idata, idata, array_size, hipMemcpyHostToDevice);

    StreamCompaction::Common::kernMapToBoolean<<<numBlocks, BLOCK_SIZE>>>(n, dev_indices, dev_idata);

    int last;
    hipMemcpy(&last, dev_indices + n-1, sizeof(int), hipMemcpyDeviceToHost);

    dv_scan(n, dev_indices);
    int streamSize;
    hipMemcpy(&streamSize, dev_indices + n-1, sizeof(int), hipMemcpyDeviceToHost);

    StreamCompaction::Common::kernScatter<<<numBlocks, BLOCK_SIZE>>>(n, dev_odata, dev_indices, dev_idata);

    hipMemcpy(odata, dev_odata, array_size, hipMemcpyDeviceToHost);

    // The kernel always copies the last elt.
    // Adjust the size to include it if desired.
    if (last == 1) {
        streamSize++;
    }

    return streamSize;
}

}
}
